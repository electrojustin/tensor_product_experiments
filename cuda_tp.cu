#include <stdint.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define LOG_NUM_THREADS 9
#define NUM_THREADS (1 << LOG_NUM_THREADS)
#define NUM_BLOCKS 1024
#define SAMPLES_PER_BLOCK 8
#define MINIBATCH_MAX_SIZE (SAMPLES_PER_BLOCK * NUM_BLOCKS)

#define BITFIELD_EXTRACT(SRC, DST, START, LEN) asm("bfe.u32 %0, %1, " #START ", " #LEN ";" : "=r"(DST) : "r"(SRC))
#define BITFIELD_EXTRACT_SIGNED(SRC, DST, START, LEN) asm("bfe.s32 %0, %1, " #START ", " #LEN ";" : "=r"(DST) : "r"(SRC))

__global__ void tensor_product_forward_kernel(
    float *__restrict__ in1_global, float *__restrict__ in2_global,
    float *__restrict__ out, float *__restrict__ cb_palette_global,
    uint32_t *__restrict__ block_jobs, uint32_t *__restrict__ block_job_sizes,
    size_t in1_size, size_t in2_size, size_t cb_palette_size, size_t out_size,
    size_t minibatch_size) {
  if (blockIdx.x * SAMPLES_PER_BLOCK >= minibatch_size ||
      threadIdx.x >= out_size) {
    return;
  }

  int batch_idx = blockIdx.x;
  in1_global += batch_idx * in1_size * SAMPLES_PER_BLOCK;
  in2_global += batch_idx * in2_size * SAMPLES_PER_BLOCK;
  out += batch_idx * out_size * SAMPLES_PER_BLOCK;

  // Copy in1, in2, and the Clebsch-Gordon coefficient palette to shared memory.
  extern __shared__ float shared_mem[];
  float* in1 = shared_mem;
  float *in2 = shared_mem + in1_size * SAMPLES_PER_BLOCK;
  float *cb_palette = shared_mem + (in1_size + in2_size) * SAMPLES_PER_BLOCK;
  for (int i = threadIdx.x; i < in1_size * SAMPLES_PER_BLOCK; i += blockDim.x) {
    in1[i] = in1_global[i];
  }
  for (int i = threadIdx.x; i < in2_size * SAMPLES_PER_BLOCK; i += blockDim.x) {
    in2[i] = in2_global[i];
  }
  for (int i = threadIdx.x; i < cb_palette_size; i += blockDim.x) {
    cb_palette[i] = cb_palette_global[i];
  }
  __syncthreads();

  // One iteration of this loop accumulates all the products needed for one
  // output row. This ensures we keep the number of writes at the absolute
  // minimum.
  for (int out_idx = threadIdx.x; out_idx < out_size; out_idx += blockDim.x) {
    // Unpack the instruction written in absolute form.
    uint32_t block_job_size = block_job_sizes[out_idx >> LOG_NUM_THREADS];
    uint32_t input_idx = block_jobs[threadIdx.x];
    int in1_idx = input_idx & 0x3FF;
    int in2_idx;
    BITFIELD_EXTRACT(input_idx, in2_idx, 10, 10);
    int cb_idx = input_idx >> 20;
    float4 acc;
    float4 acc2;
    acc.x = in1[in1_idx] * in2[in2_idx] * cb_palette[cb_idx];
    acc.y =
        in1[in1_idx + in1_size] * in2[in2_idx + in2_size] * cb_palette[cb_idx];
    acc.z = in1[in1_idx + 2 * in1_size] * in2[in2_idx + 2 * in2_size] *
            cb_palette[cb_idx];
    acc.w = in1[in1_idx + 3 * in1_size] * in2[in2_idx + 3 * in2_size] *
            cb_palette[cb_idx];
    acc2.x = in1[in1_idx + 4 * in1_size] * in2[in2_idx + 4 * in2_size] *
             cb_palette[cb_idx];
    acc2.y = in1[in1_idx + 5 * in1_size] * in2[in2_idx + 5 * in2_size] *
             cb_palette[cb_idx];
    acc2.z = in1[in1_idx + 6 * in1_size] * in2[in2_idx + 6 * in2_size] *
             cb_palette[cb_idx];
    acc2.w = in1[in1_idx + 7 * in1_size] * in2[in2_idx + 7 * in2_size] *
             cb_palette[cb_idx];

    // Decompress the delta compressed instructions.
    for (int block_job_idx = threadIdx.x + blockDim.x;
         block_job_idx < block_job_size; block_job_idx += blockDim.x) {
      input_idx = block_jobs[block_job_idx];

      int in1_delta;
      BITFIELD_EXTRACT_SIGNED(input_idx, in1_delta, 0, 5);
      in1_idx += in1_delta;
      int in2_delta;
      BITFIELD_EXTRACT(input_idx, in2_delta, 5, 1);
      in2_idx += in2_delta;
      BITFIELD_EXTRACT(input_idx, cb_idx, 6, 10);
      acc.x += in1[in1_idx] * in2[in2_idx] * cb_palette[cb_idx];
      acc.y += in1[in1_idx + in1_size] * in2[in2_idx + in2_size] *
               cb_palette[cb_idx];
      acc.z += in1[in1_idx + 2 * in1_size] * in2[in2_idx + 2 * in2_size] *
               cb_palette[cb_idx];
      acc.w += in1[in1_idx + 3 * in1_size] * in2[in2_idx + 3 * in2_size] *
               cb_palette[cb_idx];
      acc2.x += in1[in1_idx + 4 * in1_size] * in2[in2_idx + 4 * in2_size] *
                cb_palette[cb_idx];
      acc2.y += in1[in1_idx + 5 * in1_size] * in2[in2_idx + 5 * in2_size] *
                cb_palette[cb_idx];
      acc2.z += in1[in1_idx + 6 * in1_size] * in2[in2_idx + 6 * in2_size] *
                cb_palette[cb_idx];
      acc2.w += in1[in1_idx + 7 * in1_size] * in2[in2_idx + 7 * in2_size] *
                cb_palette[cb_idx];
      BITFIELD_EXTRACT_SIGNED(input_idx, in1_delta, 16, 5);
      in1_idx += in1_delta;
      BITFIELD_EXTRACT(input_idx, in2_delta, 21, 1);
      in2_idx += in2_delta;
      BITFIELD_EXTRACT(input_idx, cb_idx, 22, 10);
      acc.x += in1[in1_idx] * in2[in2_idx] * cb_palette[cb_idx];
      acc.y += in1[in1_idx + in1_size] * in2[in2_idx + in2_size] *
               cb_palette[cb_idx];
      acc.z += in1[in1_idx + 2 * in1_size] * in2[in2_idx + 2 * in2_size] *
               cb_palette[cb_idx];
      acc.w += in1[in1_idx + 3 * in1_size] * in2[in2_idx + 3 * in2_size] *
               cb_palette[cb_idx];
      acc2.x += in1[in1_idx + 4 * in1_size] * in2[in2_idx + 4 * in2_size] *
                cb_palette[cb_idx];
      acc2.y += in1[in1_idx + 5 * in1_size] * in2[in2_idx + 5 * in2_size] *
                cb_palette[cb_idx];
      acc2.z += in1[in1_idx + 6 * in1_size] * in2[in2_idx + 6 * in2_size] *
                cb_palette[cb_idx];
      acc2.w += in1[in1_idx + 7 * in1_size] * in2[in2_idx + 7 * in2_size] *
                cb_palette[cb_idx];
    }
    out[out_idx] = acc.x;
    out[out_idx + out_size] = acc.y;
    out[out_idx + 2 * out_size] = acc.z;
    out[out_idx + 3 * out_size] = acc.w;
    out[out_idx + 4 * out_size] = acc2.x;
    out[out_idx + 5 * out_size] = acc2.y;
    out[out_idx + 6 * out_size] = acc2.z;
    out[out_idx + 7 * out_size] = acc2.w;
    block_jobs += block_job_size;
  }
}

void tensor_product_forward_cuda(
    float *__restrict__ in1, float *__restrict__ in2, float *__restrict__ out,
    float *__restrict__ cb_palette, uint32_t *__restrict__ block_jobs,
    uint32_t *__restrict__ block_job_sizes, size_t in1_size, size_t in2_size,
    size_t cb_palette_size, size_t out_size, int batch_size) {
  while (batch_size > 0) {
      int minibatch_size = batch_size < MINIBATCH_MAX_SIZE ? batch_size : MINIBATCH_MAX_SIZE;
      tensor_product_forward_kernel<<<
          NUM_BLOCKS, NUM_THREADS,
          (SAMPLES_PER_BLOCK * (in1_size + in2_size) + cb_palette_size) *
              sizeof(float)>>>(in1, in2, out, cb_palette, block_jobs,
                               block_job_sizes, in1_size, in2_size,
                               cb_palette_size, out_size, minibatch_size);
      batch_size -= MINIBATCH_MAX_SIZE;
      in1 += in1_size * MINIBATCH_MAX_SIZE;
      in2 += in2_size * MINIBATCH_MAX_SIZE;
      out += out_size * MINIBATCH_MAX_SIZE;
  }
}
